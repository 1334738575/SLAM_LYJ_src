#include "hip/hip_runtime.h"
#include "CUDAProjector.h"

namespace SLAM_LYJ_CUDA
{
	__global__ void testCU(int* _as, int* _bs, int* _cs, int _sz) {
		int idx = threadIdx.x + blockDim.x * blockIdx.x;
		//int idy = threadIdx.y + blockDim.y * blockIdx.y;
		int id = idx;
		if (id >= _sz)
			return;
		_cs[id] = _as[id] + _bs[id];
	}

	void testCUDA(int* _as, int* _bs, int* _cs, int _sz)
	{
		dim3 block(32, 1);
		dim3 grid(32, 1);
		testCU << <grid, block >> > (_as, _bs, _cs, _sz);
	}
}